#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void f(const float* A, float* result, int N) { 
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i< N) {
        result[i] = sinf(cosf(logf(A[i])));
    }
}

int main() {
    int N = 10000000; 

    size_t size = N * sizeof(float);

    float* h_a = (float*)malloc(size);
    float* h_result = (float*)malloc(size);

    for (int i = 0; i < N; ++i) {
        h_a[i] = (i + 0.5)/N;
    }

    float *d_a, *d_result;
    hipMalloc(&d_a, size);
    hipMalloc(&d_result, size);

    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    f<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_result, N);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float ms = 0;
    hipEventElapsedTime(&ms, start, stop);

    hipMemcpy(h_result, d_result, size, hipMemcpyDeviceToHost);

    for (int i = 0; i < N; ++i) {
        printf("f(%f) = %f\n", h_a[i], h_result[i]);
    }

    printf("Function ( sin(cos(log(x))) ) kernel execution time: %.6f ms\n", ms);
    
    hipFree(d_a);
    hipFree(d_result);
    free(h_a);
    free(h_result);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}

